#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>

#include <iostream>

#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include "compat.h"

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) \
    CHECK_CUDA(x);     \
    CHECK_CONTIGUOUS(x)

__inline__ __device__ float WarpAllReduceMax(float val) {
    for (int mask = 1; mask < 32; mask *= 2) {
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}

__inline__ __device__ float WarpAllReduceSum(float val) {
    for (int mask = 1; mask < 32; mask *= 2) {
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

////////////////

__global__ void fastfold_softmax_fp32(float *input, float *output, long long rows, long long cols) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        float *row_input = input + row_offset * cols;
        float *row_output = output + row_offset * cols;

        float thread_max = -1 * HIP_INF_F;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            buf[i] = row_input[lane_id * cols_per_thread + i];
        }

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_output[lane_id * cols_per_thread + i] = __fdividef(buf[i], warp_sum);
        }
    }
}

__global__ void fastfold_softmax_bfp16(at::BFloat16 *input, at::BFloat16 *output, long long rows,
                                       long long cols) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {

        at::BFloat16 *row_input = input + row_offset * cols;
        at::BFloat16 *row_output = output + row_offset * cols;

        float thread_max = -1 * HIP_INF_F;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            buf[i] = static_cast<float>(row_input[lane_id * cols_per_thread + i]);
        }

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_output[lane_id * cols_per_thread + i] =
                static_cast<at::BFloat16>(__fdividef(buf[i], warp_sum));
        }
    }
}

__global__ void fastfold_softmax_grad_fp32(float *d_output, float *output, float *d_input, long long rows,
                                           long long cols) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float y_buf[32];
    float dy_buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        float *row_d_output = d_output + row_offset * cols;
        float *row_output = output + row_offset * cols;
        float *row_d_input = d_input + row_offset * cols;

        float thread_max = -1 * HIP_INF_F;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            y_buf[i] = row_output[lane_id * cols_per_thread + i];
            dy_buf[i] = row_d_output[lane_id * cols_per_thread + i];
        }

        float thread_sum = 0.f;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_sum += y_buf[i] * dy_buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_d_input[lane_id * cols_this_thread + i] = (dy_buf[i] - warp_sum) * y_buf[i];
        }
    }
}

__global__ void fastfold_softmax_grad_bfp16(at::BFloat16 *d_output, at::BFloat16 *output,
                                            at::BFloat16 *d_input, long long rows, long long cols) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float y_buf[32];
    float dy_buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        at::BFloat16 *row_d_output = d_output + row_offset * cols;
        at::BFloat16 *row_output = output + row_offset * cols;
        at::BFloat16 *row_d_input = d_input + row_offset * cols;

        float thread_max = -1 * HIP_INF_F;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            y_buf[i] = static_cast<float>(row_output[lane_id * cols_per_thread + i]);
            dy_buf[i] = static_cast<float>(row_d_output[lane_id * cols_per_thread + i]);
        }

        float thread_sum = 0.f;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_sum += y_buf[i] * dy_buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_d_input[lane_id * cols_per_thread + i] =
                static_cast<at::BFloat16>((dy_buf[i] - warp_sum) * y_buf[i]);
        }
    }
}

at::Tensor softmax(at::Tensor input, long long rows, long long cols) {
    CHECK_INPUT(input);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(input));

    at::Tensor output = at::empty_like(input);

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (input.dtype() == torch::kFloat32) {
        fastfold_softmax_fp32<<<grid, block>>>((float *)input.data_ptr(),
                                               (float *)output.data_ptr(), rows, cols);
    } else {
        fastfold_softmax_bfp16<<<grid, block>>>((at::BFloat16 *)input.data_ptr(),
                                                (at::BFloat16 *)output.data_ptr(), rows, cols);
    }

    return output;
}

at::Tensor softmax_gradient(at::Tensor d_output, at::Tensor output, long long rows, long long cols) {
    CHECK_INPUT(output);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
    at::Tensor grad_input = at::empty_like(output);

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (output.dtype() == torch::kFloat32) {
        fastfold_softmax_grad_fp32<<<grid, block>>>((float *)d_output.data_ptr(),
                                                    (float *)output.data_ptr(),
                                                    (float *)grad_input.data_ptr(), rows, cols);
    } else {
        fastfold_softmax_grad_bfp16<<<grid, block>>>(
            (at::BFloat16 *)d_output.data_ptr(), (at::BFloat16 *)output.data_ptr(),
            (at::BFloat16 *)grad_input.data_ptr(), rows, cols);
    }

    return grad_input;
}

////////////////

__global__ void fastfold_softmax_scale_mask_fp32(float *input, float *mask, float *output, long long rows,
                                                 long long cols, float scale, int head) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        float *row_input = input + row_offset * cols;
        float *row_output = output + row_offset * cols;
        float *mask_ptr = mask + ((row_offset / (head * cols)) * cols);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            if (mask_ptr[lane_id * cols_per_thread + i] == 0) {
                buf[i] = -1 * 1e9;
            } else {
                buf[i] = row_input[lane_id * cols_per_thread + i] * scale;
            }
        }

        float thread_max = -1 * HIP_INF_F;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_output[lane_id * cols_per_thread + i] = __fdividef(buf[i], warp_sum);
        }
    }
}

__global__ void fastfold_softmax_scale_mask_bfp16(at::BFloat16 *input, at::BFloat16 *mask,
                                                  at::BFloat16 *output, long long rows, long long cols,
                                                  float scale, int head) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        at::BFloat16 *row_input = input + row_offset * cols;
        at::BFloat16 *row_output = output + row_offset * cols;
        at::BFloat16 *mask_ptr = mask + ((row_offset / (head * cols)) * cols);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            if (mask_ptr[lane_id * cols_per_thread + i] == 0) {
                buf[i] = -1 * 10e9;
            } else {
                buf[i] = static_cast<float>(row_input[lane_id * cols_per_thread + i]) * scale;
            }
        }

        float thread_max = -1 * HIP_INF_F;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_output[lane_id * cols_per_thread + i] =
                static_cast<at::BFloat16>(__fdividef(buf[i], warp_sum));
        }
    }
}

at::Tensor fused_scale_mask_softmax_forward(at::Tensor input, at::Tensor mask, long long rows, long long cols,
                                            float scale) {
    CHECK_INPUT(input);
    CHECK_INPUT(mask);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
    int head = input.sizes()[2];
    at::Tensor output = at::empty_like(input);

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (input.dtype() == torch::kFloat32) {
        fastfold_softmax_scale_mask_fp32<<<grid, block>>>(
            (float *)input.data_ptr(), (float *)mask.data_ptr(), (float *)output.data_ptr(), rows,
            cols, scale, head);
    } else {
        fastfold_softmax_scale_mask_bfp16<<<grid, block>>>(
            (at::BFloat16 *)input.data_ptr(), (at::BFloat16 *)mask.data_ptr(),
            (at::BFloat16 *)output.data_ptr(), rows, cols, scale, head);
    }

    return output;
}

__global__ void fastfold_softmax_scale_mask_grad_fp32(float *d_output, float *output,
                                                      float *d_input, float *mask, long long rows,
                                                      long long cols, float scale, int head) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float y_buf[32];
    float dy_buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        float *row_d_output = d_output + row_offset * cols;
        float *row_output = output + row_offset * cols;
        float *row_d_input = d_input + row_offset * cols;
        float *mask_ptr = mask + ((row_offset / (head * cols)) * cols);

        float thread_max = -1 * HIP_INF_F;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            y_buf[i] = row_output[lane_id * cols_per_thread + i];
            dy_buf[i] = row_d_output[lane_id * cols_per_thread + i];
        }

        float thread_sum = 0.f;

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_sum += y_buf[i] * dy_buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            if (mask_ptr[lane_id * cols_per_thread + i] != 0) {
                row_d_input[lane_id * cols_per_thread + i] =
                    scale * ((dy_buf[i] - warp_sum) * y_buf[i]);
            } else {
                row_d_input = 0;
            }
        }
    }
}

__global__ void fastfold_softmax_scale_mask_grad_bfp16(at::BFloat16 *d_output, at::BFloat16 *output,
                                                       at::BFloat16 *d_input, at::BFloat16 *mask,
                                                       long long rows, long long cols, float scale, int head) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float y_buf[32];
    float dy_buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        at::BFloat16 *row_d_output = d_output + row_offset * cols;
        at::BFloat16 *row_output = output + row_offset * cols;
        at::BFloat16 *row_d_input = d_input + row_offset * cols;
        at::BFloat16 *mask_ptr = mask + ((row_offset / (head * cols)) * cols);

        float thread_max = -1 * HIP_INF_F;

        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            y_buf[i] = static_cast<float>(row_output[lane_id * cols_per_thread + i]);
            dy_buf[i] = static_cast<float>(row_d_output[lane_id * cols_per_thread + i]);
        }

        float thread_sum = 0.f;

        #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_sum += y_buf[i] * dy_buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);

        #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            if (mask_ptr[lane_id * cols_per_thread + i] != 0) {
                row_d_input[lane_id * cols_per_thread + i] =
                    static_cast<at::BFloat16>(scale * ((dy_buf[i] - warp_sum) * y_buf[i]));
            } else {
                row_d_input = 0;
            }
        }
    }
}

at::Tensor fused_scale_mask_softmax_backward(at::Tensor d_output, at::Tensor output,
                                             at::Tensor mask, long long rows, long long cols, float scale) {
    CHECK_INPUT(output);
    CHECK_INPUT(mask);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(mask));
    int head = output.sizes()[2];
    at::Tensor grad_input = at::empty_like(output);

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (output.dtype() == torch::kFloat32) {
        fastfold_softmax_scale_mask_grad_fp32<<<grid, block>>>(
            (float *)d_output.data_ptr(), (float *)output.data_ptr(),
            (float *)grad_input.data_ptr(), (float *)mask.data_ptr(), rows, cols, scale, head);
    } else {
        fastfold_softmax_scale_mask_grad_bfp16<<<grid, block>>>(
            (at::BFloat16 *)d_output.data_ptr(), (at::BFloat16 *)output.data_ptr(),
            (at::BFloat16 *)grad_input.data_ptr(), (at::BFloat16 *)mask.data_ptr(), rows, cols,
            scale, head);
    }

    return grad_input;
}

////////////////

__global__ void fastfold_softmax_scale_mask_bias_fp32(float *input, float *mask, float *bias,
                                                      float *output, long long rows, long long cols,
                                                      float scale, int head) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        float *row_input = input + row_offset * cols;
        float *row_output = output + row_offset * cols;
        float *mask_ptr = mask + ((row_offset / (head * cols)) * cols);
        float *bias_ptr = bias + ((row_offset % (head * cols)) * cols);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            if (mask_ptr[lane_id * cols_per_thread + i] == 0) {
                buf[i] = -1 * 10e9;
            } else {
                buf[i] = row_input[lane_id * cols_per_thread + i] * scale +
                        bias_ptr[lane_id * cols_per_thread + i];
            }
        }

        float thread_max = -1 * HIP_INF_F;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_output[lane_id * cols_per_thread + i] = __fdividef(buf[i], warp_sum);
        }
    }
}

__global__ void fastfold_softmax_scale_mask_bias_bfp16(at::BFloat16 *input, at::BFloat16 *mask,
                                                       at::BFloat16 *bias, at::BFloat16 *output,
                                                       long long rows, long long cols, float scale, int head) {
    int threadidx_x = threadIdx.x / 32;
    int threadidx_y = threadIdx.x % 32;
    long long row_offset = (long long)blockIdx.x * 4 + threadidx_x;
    int cols_per_thread = (cols + 31) / 32;
    int cols_this_thread = cols_per_thread;

    int last_y = (cols / cols_per_thread);

    if (threadidx_y == last_y) {
        cols_this_thread = cols - cols_per_thread * last_y;
    }
    else if (threadidx_y > last_y) {
        cols_this_thread = 0;
    }

    float buf[32];

    int lane_id = threadidx_y;

    if (row_offset < rows) {
        at::BFloat16 *row_input = input + row_offset * cols;
        at::BFloat16 *row_output = output + row_offset * cols;
        at::BFloat16 *mask_ptr = mask + ((row_offset / (head * cols)) * cols);
        at::BFloat16 *bias_ptr = bias + ((row_offset % (head * cols)) * cols);

    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            if (mask_ptr[lane_id * cols_per_thread + i] == 0) {
                buf[i] = -1 * 10e9;
            } else {
                buf[i] = static_cast<float>(row_input[lane_id * cols_per_thread + i]) * scale;
                buf[i] += static_cast<float>(bias_ptr[lane_id * cols_per_thread + i]);
            }
        }

        float thread_max = -1 * HIP_INF_F;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; i++) {
            thread_max = max(thread_max, buf[i]);
        }

        float warp_max = WarpAllReduceMax(thread_max);

        float thread_sum = 0.f;
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            buf[i] = __expf(buf[i] - warp_max);
            thread_sum += buf[i];
        }

        float warp_sum = WarpAllReduceSum(thread_sum);
    #pragma unroll
        for (int i = 0; i < cols_this_thread; ++i) {
            row_output[lane_id * cols_per_thread + i] =
                static_cast<at::BFloat16>(__fdividef(buf[i], warp_sum));
        }
    }
}

at::Tensor fused_scale_mask_bias_softmax_forward(at::Tensor input, at::Tensor mask, at::Tensor bias,
                                                 long long rows, long long cols, float scale) {
    CHECK_INPUT(input);
    CHECK_INPUT(mask);
    CHECK_INPUT(bias);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(input));
    int head = input.sizes()[2];
    at::Tensor output = at::empty_like(input);

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (input.dtype() == torch::kFloat32) {
        fastfold_softmax_scale_mask_bias_fp32<<<grid, block>>>(
            (float *)input.data_ptr(), (float *)mask.data_ptr(), (float *)bias.data_ptr(),
            (float *)output.data_ptr(), rows, cols, scale, head);
    } else {
        fastfold_softmax_scale_mask_bias_bfp16<<<grid, block>>>(
            (at::BFloat16 *)input.data_ptr(), (at::BFloat16 *)mask.data_ptr(),
            (at::BFloat16 *)bias.data_ptr(), (at::BFloat16 *)output.data_ptr(), rows, cols, scale,
            head);
    }

    return output;
}

at::Tensor fused_scale_mask_bias_softmax_backward(at::Tensor d_output, at::Tensor output,
                                                  at::Tensor mask, at::Tensor bias, long long rows,
                                                  long long cols, float scale) {
    CHECK_INPUT(output);
    CHECK_INPUT(mask);
    const at::cuda::OptionalCUDAGuard device_guard(device_of(mask));
    int head = output.sizes()[2];
    at::Tensor grad_input = at::empty_like(output);

    int grid = (rows + 3) / 4;
    dim3 block(128);

    if (output.dtype() == torch::kFloat32) {
        fastfold_softmax_scale_mask_grad_fp32<<<grid, block>>>(
            (float *)d_output.data_ptr(), (float *)output.data_ptr(),
            (float *)grad_input.data_ptr(), (float *)mask.data_ptr(), rows, cols, scale, head);
    } else {
        fastfold_softmax_scale_mask_grad_bfp16<<<grid, block>>>(
            (at::BFloat16 *)d_output.data_ptr(), (at::BFloat16 *)output.data_ptr(),
            (at::BFloat16 *)grad_input.data_ptr(), (at::BFloat16 *)mask.data_ptr(), rows, cols,
            scale, head);
    }

    return grad_input;
}
